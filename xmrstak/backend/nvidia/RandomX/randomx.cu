/*
Copyright (c) 2019 SChernykh

This file is part of RandomX CUDA.

RandomX CUDA is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

RandomX CUDA is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with RandomX CUDA.  If not, see<http://www.gnu.org/licenses/>.
*/

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdint.h>
//#include "../CryptoNight_constants.h"
#include "xmrstak/backend/nvidia/nvcc_code/cryptonight.hpp"
#include "xmrstak/backend/cpu/crypto/cryptonight_1.h"
#include "../nvcc_code/cuda_device.hpp"
//#include "../workers/Workers.h"

void randomx_prepare(nvid_ctx *ctx, const uint8_t* seed_hash, const xmrstak_algo& miner_algo, uint32_t batch_size)
{
	printf("NVIDIA: start prep dataset\n");
	//constexpr size_t RANDOMX_DATASET_MAX_SIZE = 2181038080llu;
    const size_t dataset_size = getRandomXDatasetSize();
    if (!ctx->d_rx_dataset) {
		printf("NVIDIA: create dataset\n");
        CUDA_CHECK(ctx->device_id, hipMalloc(&ctx->d_rx_dataset, dataset_size));
    }
    if (!ctx->d_long_state) {
        ctx->d_scratchpads_size = batch_size * miner_algo.Mem();
        CUDA_CHECK(ctx->device_id, hipMalloc(&ctx->d_long_state, ctx->d_scratchpads_size));
    }
    if (!ctx->d_rx_hashes) {
        CUDA_CHECK(ctx->device_id, hipMalloc(&ctx->d_rx_hashes, batch_size * 64));
    }
    if (!ctx->d_rx_entropy) {
        CUDA_CHECK(ctx->device_id, hipMalloc(&ctx->d_rx_entropy, batch_size * (128 + 2560)));
    }
    if (!ctx->d_rx_vm_states) {
        CUDA_CHECK(ctx->device_id, hipMalloc(&ctx->d_rx_vm_states, batch_size * 2560));
    }
    if (!ctx->d_rx_rounding) {
        CUDA_CHECK(ctx->device_id, hipMalloc(&ctx->d_rx_rounding, batch_size * sizeof(uint32_t)));
    }

	///@todo check if variant remove is ok
    if ((memcmp(ctx->rx_dataset_seedhash, seed_hash, sizeof(ctx->rx_dataset_seedhash)) != 0) /*|| (ctx->rx_variant != variant)*/) {
        memcpy(ctx->rx_dataset_seedhash, seed_hash, sizeof(ctx->rx_dataset_seedhash));
        //ctx->rx_variant = variant;
		printf("NVIDIA: h2d dataset %lu\n",dataset_size);
        CUDA_CHECK(ctx->device_id, hipMemcpy(ctx->d_rx_dataset, getRandomXDataset(), dataset_size, hipMemcpyHostToDevice));
    }
	printf("NVIDIA: prep finished dataset\n");
}
